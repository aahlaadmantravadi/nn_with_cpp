#include "hip/hip_runtime.h"
// =============================================================================
// File: src/backend/gpu/GpuOps.cu
// =============================================================================
//
// Description: Implements the CUDA kernels for high-performance GPU
//              computation. This includes a tiled matrix multiplication kernel
//              for efficiency and simpler element-wise kernels.
//
// =============================================================================

#include "backend/gpu/GpuOps.cuh"
#include "nn/Tensor.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

// --- CUDA Kernel Definitions ---

// Define tile width for shared memory optimization in matmul
#define TILE_WIDTH 32

// CUDA Kernel for Tiled Matrix Multiplication: C = A * B
__global__ void matmulKernel(const float* A, const float* B, float* C, int m, int k, int n) {
    // Shared memory for tiles of A and B
    __shared__ float sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sB[TILE_WIDTH][TILE_WIDTH];

    // Thread indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Global row and column for this thread's element in C
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float Cvalue = 0.0f;

    // Loop over the tiles of A and B required to compute the C element
    for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        // Load tile of A into shared memory
        if (row < m && (t * TILE_WIDTH + tx) < k) {
            sA[ty][tx] = A[row * k + (t * TILE_WIDTH + tx)];
        } else {
            sA[ty][tx] = 0.0f;
        }

        // Load tile of B into shared memory
        if (col < n && (t * TILE_WIDTH + ty) < k) {
            sB[ty][tx] = B[(t * TILE_WIDTH + ty) * n + col];
        } else {
            sB[ty][tx] = 0.0f;
        }

        // Synchronize to make sure the tiles are loaded
        __syncthreads();

        // Multiply the two tiles and accumulate the result
        for (int i = 0; i < TILE_WIDTH; ++i) {
            Cvalue += sA[ty][i] * sB[i][tx];
        }
        
        // Synchronize to make sure all threads are done with the current tile
        __syncthreads();
    }

    // Write the final result to global memory
    if (row < m && col < n) {
        C[row * n + col] = Cvalue;
    }
}

// CUDA Kernel for element-wise addition
__global__ void addKernel(const float* A, const float* B, float* C, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        C[idx] = A[idx] + B[idx];
    }
}

// CUDA Kernel for ReLU activation
__global__ void reluKernel(float* A, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        A[idx] = fmaxf(0.0f, A[idx]);
    }
}


// --- C++ Wrapper Functions (Implementation of GpuOps) ---

void GpuOps::matmul(const Tensor& A, const Tensor& B, Tensor& C) {
    size_t m = A.getRows();
    size_t k = A.getCols();
    size_t n = B.getCols();

    if (k != B.getRows()) {
        throw std::invalid_argument("Matrix dimensions are incompatible for multiplication.");
    }

    // Get raw GPU data pointers
    const float* a_data = A.getGpuData();
    const float* b_data = B.getGpuData();
    float* c_data = C.getGpuData();

    if (!a_data || !b_data || !c_data) {
        throw std::runtime_error("matmul: One or more tensors are not on the GPU.");
    }

    // Define grid and block dimensions for the kernel launch
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 numBlocks((n + TILE_WIDTH - 1) / TILE_WIDTH, (m + TILE_WIDTH - 1) / TILE_WIDTH);

    // Launch the kernel
    matmulKernel<<<numBlocks, threadsPerBlock>>>(a_data, b_data, c_data, m, k, n);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA matmul kernel launch failed: ") + hipGetErrorString(err));
    }
}

void GpuOps::add(const Tensor& A, const Tensor& B, Tensor& C) {
    if (A.getSize() != B.getSize() || A.getSize() != C.getSize()) {
        throw std::invalid_argument("Tensors must have the same size for addition.");
    }

    const float* a_data = A.getGpuData();
    const float* b_data = B.getGpuData();
    float* c_data = C.getGpuData();

    if (!a_data || !b_data || !c_data) {
        throw std::runtime_error("add: One or more tensors are not on the GPU.");
    }
    
    size_t size = A.getSize();
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    addKernel<<<numBlocks, threadsPerBlock>>>(a_data, b_data, c_data, size);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA add kernel launch failed: ") + hipGetErrorString(err));
    }
}

void GpuOps::relu(Tensor& A) {
    float* a_data = A.getGpuData();
    if (!a_data) {
        throw std::runtime_error("relu: Tensor is not on the GPU.");
    }

    size_t size = A.getSize();
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    reluKernel<<<numBlocks, threadsPerBlock>>>(a_data, size);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA relu kernel launch failed: ") + hipGetErrorString(err));
    }
}
